#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

__global__ void CountSort(int*, int*, int, int);

__host__ void counting_sort(int* arr, int size, int max_val)
{
	int block_num = 5;
	int thread_num_per_block = 5;
	uint64_t histo_size = sizeof(int)*max_val;
	printf("size: %d\n", size);
	printf("max_val: %d\n", max_val);
	printf("block_num: %d\n", block_num);
	printf("thread_per_block: %d\n", thread_num_per_block);

	int* darr;
	hipMalloc(&darr, sizeof(int)*size);
	hipMemcpy(darr, arr, sizeof(int)*size, hipMemcpyHostToDevice); 

	int* dhisto;
	hipMalloc(&dhisto, histo_size);
	hipMemset(dhisto, 0, histo_size);

	printf("countsort start\n");
	CountSort<<<block_num, thread_num_per_block>>>(darr, dhisto, size, max_val);
	printf("countsort end\n");
	
	int* histo = (int*)calloc(max_val, sizeof(int)); 
	hipMemcpy(histo, dhisto, sizeof(int)*max_val, hipMemcpyDeviceToHost);
	hipMemcpy(arr, darr, sizeof(int)*size, hipMemcpyDeviceToHost);
	
	
	//printf("total_cnt: %d\n", histo[max_val-1]);
	
	/*
	int idx = 0;
	for(int i=0; i<max_val; i++) {
		for(int j=0; j<histo[i]; j++) {
			arr[idx++] = i;
		}
	}
	*/
	//cudaFree(dhisto);
	//cudaFree(darr);
	//free(histo);
}

__global__ void CountSort(int* darr, int* dhisto, int size, int max_val) {

	uint64_t thread_per_block = blockDim.x;
	uint64_t total_block = gridDim.x;
	uint64_t bid = blockIdx.x;
	uint64_t tid = threadIdx.x;
	uint64_t size_per_block, bstart, size_per_thread, start, end;

	// update histogram	in each block
	if(size % total_block != 0 && bid == total_block - 1) {
		size_per_block = size / total_block + size % total_block;
		bstart = bid * (size / total_block);
		size_per_thread = size_per_block / thread_per_block;
		start = bstart + tid * size_per_thread;
		end = start + size_per_thread;
		if(size_per_block % thread_per_block != 0 && 
				tid == thread_per_block - 1) {
			end += size_per_block % thread_per_block;
		}
	}
	else {
		size_per_block = size / total_block;
		bstart = bid * size_per_block;	
		size_per_thread = size_per_block / thread_per_block;
		start = bstart + tid * size_per_thread;
		end = start + size_per_thread;
		if(size_per_block % thread_per_block != 0 && tid == thread_per_block - 1) {
			end += size_per_block % thread_per_block;
		}
	}
	for(uint64_t i=start; i<end; i++) {
		atomicAdd(&dhisto[darr[i]], 1);
	}
	__syncthreads();
	

	if(bid == 0 && tid == 0 ) {
		for(int i=1; i<max_val; i++) {
			dhisto[i] += dhisto[i-1];
		}
		printf("%d\n", dhisto[max_val-1]);
	}

	__syncthreads();

	size_per_block = max_val / total_block;
	bstart = bid * size_per_block;
	size_per_thread = size_per_block / thread_per_block;
	start = bstart + tid * size_per_thread;
	end = start + size_per_thread;

	for(int i=start; i<end; i++) {
		if(i == 0) {
			for(int j=0; j<dhisto[0]; j++)
				darr[j] = i;
		}
		else {
			for(int j=dhisto[i-1]; j<dhisto[i]; j++) {
				darr[j] = i;	
			}
		}
	}
	
	__syncthreads();
}
